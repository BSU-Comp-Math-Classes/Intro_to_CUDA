//Just your regular Hello World file
// to be compiled with nvcc rather than gcc


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {
  printf("Hello World from GPU, thread %d of block %d \n",threadIdx.x,blockIdx.x);
}


int main(void) {
  printf("Hello World from CPU!\n");

  helloFromGPU<<<2, 7>>>();
  hipDeviceReset();

  return 0;
}
